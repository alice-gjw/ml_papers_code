// addition of a+b, fp32 v fp16, time benchmarking, Tesla L4

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <iomanip>

__global__ void fp32_add_kernel(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void fp16_add_kernel(half* a, half* b, half* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = __hadd(a[idx], b[idx]);
    }
}

float benchmark_fp32(int n, int num_runs) {
    float *d_a32, *d_b32, *d_c32;
    hipMalloc(&d_a32, n * sizeof(float));
    hipMalloc(&d_b32, n * sizeof(float));
    hipMalloc(&d_c32, n * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        fp32_add_kernel<<<gridSize, blockSize>>>(d_a32, d_b32, d_c32, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float fp32_time;
    hipEventElapsedTime(&fp32_time, start, stop);

    hipFree(d_a32);
    hipFree(d_b32);
    hipFree(d_c32);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return fp32_time;
}

float benchmark_fp16(int n, int num_runs) {
    half *d_a16, *d_b16, *d_c16;
    hipMalloc(&d_a16, n * sizeof(half));
    hipMalloc(&d_b16, n * sizeof(half));
    hipMalloc(&d_c16, n * sizeof(half));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEventRecord(start);
    for (int i = 0; i < num_runs; i++) {
        fp16_add_kernel<<<gridSize, blockSize>>>(d_a16, d_b16, d_c16, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float fp16_time;
    hipEventElapsedTime(&fp16_time, start, stop);

    hipFree(d_a16);
    hipFree(d_b16);
    hipFree(d_c16);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return fp16_time;
}

int main() {
    const int n = 1024 * 1024;
    const int num_runs = 100;

    float fp32_time = benchmark_fp32(n, num_runs);
    float fp16_time = benchmark_fp16(n, num_runs);

    std::cout << "FP32 time: " << fp32_time << " ms" << std::endl;
    std::cout << "FP16 time: " << fp16_time << " ms" << std::endl;
    std::cout << "Speedup: " << std::fixed << std::setprecision(2)
                << fp32_time / fp16_time << "x" << std::endl;

    return 0;
}

/*

nvidia-smi
nvcc --version
nvidia-smi --query-gpu=name,compute_cap --format=csv,noheader
%%writefile fp3216_bm.cu
nvcc -O3 -arch=sm_75 -o fp3216_bm fp3216_bm.cu
./fp3216_bm

FP32 time: 5.27155 ms
FP16 time: 3.87587 ms
Speedup: 1.36x

*/
